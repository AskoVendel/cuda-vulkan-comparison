#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define _CRT_SECURE_NO_WARNINGS

#include <cmath>
#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

using namespace std;

#define CHANNELS 3

void average_filter(int N, float* coeff) {
    for (int i = 0; i <= N; i++)
        for (int j = 0; j <= N; j++)
            coeff[i * (N + 1) + j] = 1;
}

__global__ void lowpass_filter_kernel(unsigned char* d_data, int width, int height, unsigned char* d_out, int N, float* d_coeff, float divider) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return; // Bounds check

    for (int c = 0; c < 3; c++) {  // RGB
        int index = (y * width + x) * 3 + c;

        
        if (y < N || x < N || y >= height - N || x >= width - N) {
            d_out[index] = d_data[index];
            continue;
        }

        float sum = 0.0f;
        for (int a = -N; a <= N; a++) {
            for (int b = -N; b <= N; b++) {
                int neighbor_idx = ((y + b) * width + (x + a)) * 3 + c;
                sum += d_data[neighbor_idx] * d_coeff[(abs(a) * (N + 1)) + abs(b)];
            }
        }
        d_out[index] = static_cast<unsigned char>(sum / divider);
    }
}

void sharpening_filter(int N, float* coeff) {
    int w = 2 * N + 1; // filtri tegelik laius

    float dist = sqrt(2 * N * N);
    float mul = dist / asin(1);
    //printf("%.3f %.3f %.3f\n", dist, asin(1), mul);

    for (int i = 0; i <= N; i++)
        for (int j = 0; j <= N; j++) {
            coeff[i * (N + 1) + j] = -cos(sqrt(i * i + j * j) / mul);
            //printf("%d,%d(%.3f): %.3f\n", i, j, sqrt(i * i + j * j), coeff[i * (N + 1) + j]);
        }
    coeff[0] = 0;
    float divider = 0;
    for (int i = -N; i <= N; i++)
        for (int j = -N; j <= N; j++)
            divider = divider + coeff[abs(i) * (N + 1) + abs(j)];
    coeff[0] = -divider;
    //printf("%.3f\n", divider);
}

__global__ void highpass_filter_kernel(unsigned char* d_data, int width, int height,
    unsigned char* d_out, int N, float* d_coeff, float sharp_mul) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;  // Bounds check

    for (int c = 0; c < 3; c++) {  // RGB
        int index = (y * width + x) * 3 + c;

        
        if (y < N || x < N || y >= height - N || x >= width - N) {
            d_out[index] = d_data[index];
            continue;
        }

        
        float sum = 0;
        for (int a = -N; a <= N; a++) {
            for (int b = -N; b <= N; b++) {
                int neighbor_idx = ((y + b) * width + (x + a)) * 3 + c;
                sum += d_data[neighbor_idx] * d_coeff[abs(a) * (N + 1) + abs(b)];
            }
        }

        
        float new_value = d_data[index] + sum * sharp_mul;
        d_out[index] = max(0, min(255, (int)new_value));  // Clamp to [0,255]
    }
}

void process_noise(void) {
    int x;
    int y;
    int n;
    int ok;

    unsigned char* d_data;
    float* d_c1;
    unsigned char* d_out1;
    float* d_c4;
    unsigned char* d_out2;

    ok = stbi_info("noise.bmp", &x, &y, &n);

    cout << "OK?: " << ok << "\n";
    cout << "width: " << x << "\n";
    cout << "height: " << y << "\n";
    cout << "bytes per pixel: " << n << "\n";
    cout << "-------------------------" << "\n";

    unsigned char* data = stbi_load("noise.bmp", &x, &y, &n, 3);

    unsigned char* out = new unsigned char[x * y * 3];
    unsigned char* out1 = new unsigned char[x * y * 3];
    unsigned char* out2 = new unsigned char[x * y * 3];
    unsigned char* out3 = new unsigned char[x * y * 3];

    // Allocate Data memory on device
    hipError_t errDataMalloc = hipMalloc((void**)&d_data, (sizeof(unsigned char) * y * x * CHANNELS));
    if (errDataMalloc != hipSuccess) {
        std::cerr << "CUDA DataMalloc failed: " << hipGetErrorString(errDataMalloc) << std::endl;
    }

    // Data to device memory
    hipError_t errDataMemCpy = hipMemcpy(d_data, data, (sizeof(unsigned char) * y * x * CHANNELS), hipMemcpyHostToDevice);
    if (errDataMemCpy != hipSuccess) {
        std::cerr << "CUDA DataMemCpy failed: " << hipGetErrorString(errDataMemCpy) << std::endl;
    }

    n = 4;
    float* c1 = new float[(n + 1) * (n + 1)];
    average_filter(n, c1);

    // Allocate c1 memory on device
    hipError_t errC1Malloc = hipMalloc((void**)&d_c1, (sizeof(float) * (n + 1) * (n + 1)));
    if (errC1Malloc != hipSuccess) {
        std::cerr << "CUDA DataMalloc failed: " << hipGetErrorString(errDataMalloc) << std::endl;
    }

    // c1 to device memory
    hipError_t errC1MemCpy = hipMemcpy(d_c1, c1, (sizeof(float) * (n + 1) * (n + 1)), hipMemcpyHostToDevice);
    if (errC1MemCpy != hipSuccess) {
        std::cerr << "CUDA DataMemCpy failed: " << hipGetErrorString(errDataMemCpy) << std::endl;
    }

    // Allocate out1 memory on device
    hipError_t errOut1Malloc = hipMalloc((void**)&d_out1, (sizeof(unsigned char) * y * x * CHANNELS));
    if (errOut1Malloc != hipSuccess) {
        std::cerr << "CUDA DataMalloc failed: " << hipGetErrorString(errDataMalloc) << std::endl;
    }

    // out1 to device memory
    hipError_t errOut1MemCpy = hipMemcpy(d_out1, out1, (sizeof(unsigned char) * y * x * CHANNELS), hipMemcpyHostToDevice);
    if (errOut1MemCpy != hipSuccess) {
        std::cerr << "CUDA DataMemCpy failed: " << hipGetErrorString(errDataMemCpy) << std::endl;
    }

    float divider = 0;
    for (int i = -n; i <= n; i++)
        for (int j = -n; j <= n; j++)
            divider += c1[abs(i) * (n + 1) + abs(j)];
    
    dim3 blockSize(16, 16);
    dim3 gridSize((x + blockSize.x - 1) / blockSize.x, (y + blockSize.y - 1) / blockSize.y);
    lowpass_filter_kernel << <gridSize, blockSize >> > (d_data, x, y, d_out1, n, d_c1, divider);
    hipDeviceSynchronize();

    // out1 back to host memory
    hipError_t errOutMemCpy = hipMemcpy(out1, d_out1, (sizeof(unsigned char) * y * x * CHANNELS), hipMemcpyDeviceToHost);
    if (errOutMemCpy != hipSuccess) {
        std::cerr << "CUDA DataMemCpy failed: " << hipGetErrorString(errDataMemCpy) << std::endl;
    }

    stbi_write_bmp("noise_blur1.bmp", x, y, CHANNELS, out1);

    n = 2;
    float* c4 = new float[(n + 1) * (n + 1)];
    sharpening_filter(n, c4);

    // Allocate c4 memory on device
    hipError_t errC4Malloc = hipMalloc((void**)&d_c4, (sizeof(float) * (n + 1) * (n + 1)));
    if (errC4Malloc != hipSuccess) {
        std::cerr << "CUDA DataMalloc failed: " << hipGetErrorString(errDataMalloc) << std::endl;
    }

    // c4 to device memory
    hipError_t errC4MemCpy = hipMemcpy(d_c4, c1, (sizeof(float) * (n + 1) * (n + 1)), hipMemcpyHostToDevice);
    if (errC4MemCpy != hipSuccess) {
        std::cerr << "CUDA DataMemCpy failed: " << hipGetErrorString(errDataMemCpy) << std::endl;
    }

    // Allocate out2 memory on device
    hipError_t errOut2Malloc = hipMalloc((void**)&d_out2, (sizeof(unsigned char) * y * x * CHANNELS));
    if (errOut2Malloc != hipSuccess) {
        std::cerr << "CUDA DataMalloc failed: " << hipGetErrorString(errDataMalloc) << std::endl;
    }


    highpass_filter_kernel << <gridSize, blockSize >> > (d_out1, x, y, d_out2, n, d_c4, 1.2);
    hipDeviceSynchronize();

    // out1 back to host memory
    hipError_t errOut2MemCpy = hipMemcpy(out2, d_out2, (sizeof(unsigned char) * y * x * CHANNELS), hipMemcpyDeviceToHost);
    if (errOut2MemCpy != hipSuccess) {
        std::cerr << "CUDA DataMemCpy failed: " << hipGetErrorString(errDataMemCpy) << std::endl;
    }

    stbi_write_bmp("noise_sharp1.bmp", x, y, CHANNELS, out2);

}


int main() {
    process_noise();

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
